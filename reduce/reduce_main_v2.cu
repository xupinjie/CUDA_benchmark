#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include "xmath.h"

template<typename T, int BLOCK_SIZE, int WARP_SIZE>
__global__ void reduce_shuffle(
    const T* src,
    int len,
    T *dst)
{
    const int tid = threadIdx.x;
    const int bid = blockIdx.x;
    const int lid = threadIdx.x % WARP_SIZE;
    const int wid = threadIdx.x / WARP_SIZE;
    const int warp_num = BLOCK_SIZE / WARP_SIZE;

    int idx = bid * BLOCK_SIZE + tid;
    T sum = (T)0;
    if (idx < len) sum = src[idx];
    sum += __shfl_down_sync(0xffffffff, sum, 16);
    sum += __shfl_down_sync(0xffffffff, sum, 8);
    sum += __shfl_down_sync(0xffffffff, sum, 4);
    sum += __shfl_down_sync(0xffffffff, sum, 2);
    sum += __shfl_down_sync(0xffffffff, sum, 1);

    __shared__ T smem[warp_num];
    if (lid == 0) smem[wid] = sum;
    __syncthreads();

    T val = (T)0;
    if (tid < warp_num) {
        val = smem[tid];
        val += __shfl_down_sync(0xffffffff, val, 16);
        val += __shfl_down_sync(0xffffffff, val, 8);
        val += __shfl_down_sync(0xffffffff, val, 4);
        val += __shfl_down_sync(0xffffffff, val, 2);
        val += __shfl_down_sync(0xffffffff, val, 1);
    }

    if (tid == 0) {
        dst[bid] = val;
    }
}

























// template<typename T, int BLOCK_SIZE, int WARP_SIZE>
// __global__ void reduce_shuffle(
//     const T* src,
//     int len,
//     T* dst)
// {
//     const int tid = threadIdx.x;
//     const int bid = blockIdx.x;
//     const int lid = threadIdx.x % WARP_SIZE;
//     const int wid = threadIdx.x / WARP_SIZE;
//     const int warp_num = BLOCK_SIZE / WARP_SIZE;

//     int idx = bid * BLOCK_SIZE + tid;
//     T sum = (T)0;
//     if (idx < len) sum = src[idx];

//     sum += __shfl_down_sync(0xffffffff, sum, 16);
//     sum += __shfl_down_sync(0xffffffff, sum, 8);
//     sum += __shfl_down_sync(0xffffffff, sum, 4);
//     sum += __shfl_down_sync(0xffffffff, sum, 2);
//     sum += __shfl_down_sync(0xffffffff, sum, 1);

//     __shared__ T smem[warp_num];
//     if (lid == 0) smem[wid] = sum;
//     __syncthreads();

//     T val = 0.;
//     if (tid < warp_num) {
//         val = smem[tid];
//         val += __shfl_down_sync(0xffffffff, val, 16);
//         val += __shfl_down_sync(0xffffffff, val, 8);
//         val += __shfl_down_sync(0xffffffff, val, 4);
//         val += __shfl_down_sync(0xffffffff, val, 2);
//         val += __shfl_down_sync(0xffffffff, val, 1);
//     }
//     if (tid == 0) dst[bid] = val;
// }

// template<int BLOCK_SIZE, int WARP_SIZE>
// __global__ void reduce_shuffle(
//     const float* inData, 
//     int len,
//     float *res)
// {
//     int tid = threadIdx.x;
//     int bid = blockIdx.x;
//     int wid = threadIdx.x / WARP_SIZE;
//     int lid = threadIdx.x % WARP_SIZE;
//     int idx = bid * BLOCK_SIZE + tid;

//     const int warp_num = BLOCK_SIZE / WARP_SIZE;

//     float sum = 0;
//     if (idx < len) sum = inData[idx];

//     sum += __shfl_down_sync(0xffffffff, sum, 16);
//     sum += __shfl_down_sync(0xffffffff, sum, 8);
//     sum += __shfl_down_sync(0xffffffff, sum, 4);
//     sum += __shfl_down_sync(0xffffffff, sum, 2);
//     sum += __shfl_down_sync(0xffffffff, sum, 1);

//     __shared__ float smem[warp_num];
//     if(lid == 0) smem[wid] = sum;
//     __syncthreads();

//     float val = 0.;
//     if (tid < warp_num) {
//         val = smem[tid];
//         val += __shfl_down_sync(0xffffffff, val, 16);
//         val += __shfl_down_sync(0xffffffff, val, 8);
//         val += __shfl_down_sync(0xffffffff, val, 4);
//         val += __shfl_down_sync(0xffffffff, val, 2);
//         val += __shfl_down_sync(0xffffffff, val, 1);
//     }

//     if (tid == 0) {
//         res[bid] = val;
//     }
// }
 
static int reduce_sum(float *inData, int len, float *res) 
{
    const int block_size     = 512;
    const int elem_per_block = block_size * 1;
    const int block_num      = ceil(len, elem_per_block);
    const int warp_size      = 32;

    // printf("block_size:%d elem_per_block:%d block_num:%d\n", block_size, elem_per_block, block_num);

    float *resBufferDevice;
    hipMalloc(&resBufferDevice, sizeof(float) * block_num);
    hipMemset(resBufferDevice, '\0', sizeof(float) * block_num);

    reduce_shuffle<float ,block_size, warp_size><<<block_num, block_size>>>(inData, len, resBufferDevice);
    reduce_shuffle<float, block_size, warp_size><<<1, block_size>>>(resBufferDevice, block_num, res);
    
    hipDeviceSynchronize();

    hipFree(resBufferDevice);

    return 0;
}

/////////////////////////////////////////////////////////////
static void init_data(float *inData, int DSIZE)
{
    for (int i = 0; i < DSIZE; i++) {
        // inData[i] = (float)1;
        inData[i] = (float)i;
    }
}

static void reduce_sum_host(float *inData, int DSIZE, float &res)
{
    res = 0;
    for (int i = 0; i < DSIZE; i++) {
        res += inData[i];
    }
}

int main()
{
    const int warm   = 5;
    const int trials = 10;

    hipEvent_t start, stop;
    float elapsedTime = 0.0;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    const int DSIZE = 51200;
    float *inData = (float *)malloc(sizeof(float) * DSIZE);

    /* init inData */
    init_data(inData, DSIZE);
    
    /* get ref */
    float ref = 0.;
    reduce_sum_host(inData, DSIZE, ref);

    float *inDataDevice;
    hipMalloc(&inDataDevice, sizeof(float) * DSIZE);
    hipMemcpy(inDataDevice, inData, sizeof(float)*DSIZE, hipMemcpyHostToDevice);

    float *res = (float *)malloc(1*sizeof(float));
    float *resDevice;
    hipMalloc(&resDevice, 1*sizeof(float));

    for (int i = 0; i < warm; i++) {
        reduce_sum(inDataDevice, DSIZE, resDevice);
    }

    hipEventRecord(start, 0);
    for (int i = 0; i < trials; i++) {
        reduce_sum(inDataDevice, DSIZE, resDevice);
    }
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("hipEventElapsedTime: %f\n", elapsedTime/trials);

    hipMemcpy(res, resDevice, 1*sizeof(float), hipMemcpyDeviceToHost);

    printf("ref:%f res:%f\n", ref, *res);

    free(inData);
    free(res);
    hipFree(inDataDevice);
    hipFree(resDevice);

    return 0;
}
